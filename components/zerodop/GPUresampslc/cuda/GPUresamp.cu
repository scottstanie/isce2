//
// Author: Joshua Cohen
// Copyright 2017
//

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include <stdio.h>
#include <sys/time.h>

#define SINC_SUB 8192
#define SINC_LEN 8
#define SINC_HALF (SINC_LEN/2)
#define SINC_ONE (SINC_LEN+1)


#define IDX1D(i,j,w) (((i)*(w))+(j))
#define modulo_f(a,b) fmod(fmod(a,b)+(b),(b))


struct InputData {
    hipFloatComplex *imgIn;
    hipFloatComplex *imgOut;
    float *residAz;
    float *residRg;
    double *azOffPoly;
    double *rgOffPoly;
    double *dopPoly;
    double *azCarrierPoly;
    double *rgCarrierPoly;
    float *fintp;
};

__constant__ double ind[6];
__constant__ int ini[8];

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//              GPU Helper Functions
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

// Data usage: 8 floats/pointers, 2 ints   --      72 bytes/call
__device__ double evalPolyAt(double *polyArr, double azi, double rng) {
    // C-style eval method of Poly2d (adjusted to work with the array-format Poly2d where:
    //  polyArr[0] = azimuthOrder
    //  polyArr[1] = rangeOrder
    //  polyArr[2] = azimuthMean
    //  polyArr[3] = rangeMean
    //  polyArr[4] = azimuthNorm
    //  polyArr[5] = rangeNorm
    //  polyArr[6...] = coeffs (len ([0]+1)*([1]+1))
    // Therefore we can guarantee that polyArr has at least 7 elements, and intuitively stores its own length using the orders

    double val, scalex, scaley, xval, yval;
    int i, j;
    val = 0.;
    scaley = 1.;
    xval = (rng - polyArr[3]) / polyArr[5];
    yval = (azi - polyArr[2]) / polyArr[4];
    for (i=0; i<=polyArr[0]; i++,scaley*=yval) {
        scalex = 1.;
        for (j=0; j<=polyArr[1]; j++,scalex*=xval) {
            val += scalex * scaley * polyArr[IDX1D(i,j,int(polyArr[1])+1)+6];
        }
    }
    return val;
}

__global__ void removeCarrier(struct InputData inData) {
    // remove the carriers from input slc
    // thread id, as the pixel index for the input image
    int pix = blockDim.x * blockIdx.x + threadIdx.x;
    // check the thread range
    // ini[0] - inLength
    // ini[1] - inWidth
    if(pix >= ini[0]*ini[1])
        return;

    // get pixel location along azimuth/range
    int idxi = pix/ini[1];
    int idxj = pix%ini[1];

    // the poly uses fortran 1-indexing
    double r_i = idxi +1;
    double r_j = idxj +1;
    // get the phase shift due to carriers
    double ph =  evalPolyAt(inData.rgCarrierPoly, r_i, r_j) +
        evalPolyAt(inData.azCarrierPoly, r_i, r_j);
    ph = modulo_f(ph, 2.*M_PI);
    // remove the phase shift from the data
    hipFloatComplex cval = hipCmulf(inData.imgIn[pix], make_hipFloatComplex(cosf(ph), -sinf(ph)));
    // assign the new value
    inData.imgIn[pix] = cval;
    // all done
}

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//              GPU Main Kernel
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

// Data Usage: 15 pointers/floats, 5 ints, 1 bool      --      144 bytes/call (assuming 1 bool ==> 1 int)
//             Add call to sinfc_interp (100 bytes/call) --      244 bytes/call (for funsies let's assume ~250 bytes/call)
// NOTE: We ignore calls to evalPolyAt sinfce they have less
//       data usage and therefore do not really matter for
//       max data usage
__global__ void GPUResamp(struct InputData inData) {
    // Main GPU ResampSlc kernel, slightly modified from original algorithm to save significant space

    int pix = blockDim.x * blockIdx.x + threadIdx.x;

    // check within outWidth*LINES_PER_TILE
    if (pix >= (ini[2] * ini[6]))
        return;

    // index along row/azimuth
    int idxi = (pix / ini[2]) + ini[4];
    // index along width/range
    int idxj = (pix % ini[2]);

    // offset
    // note that the polys use 1-indexing in Fortran code
    double ao = evalPolyAt(inData.azOffPoly, idxi+1, idxj+1) + inData.residAz[pix];
    double ro = evalPolyAt(inData.rgOffPoly, idxi+1, idxj+1) + inData.residRg[pix];

    // azimuth coordinate
    int ka = floor(idxi + ao);
    double fraca = idxi + ao - ka;
    // range coordinate
    int kr = floor(idxj + ro);
    double fracr = idxj + ro - kr;
    // check whether the pixel is out of the interpolation region
    if ((ka < SINC_HALF) || ( ka >= (ini[0]-SINC_HALF))
        || (kr < SINC_HALF) || (kr >= (ini[1]-SINC_HALF)))
    {
        // out of range
        inData.imgOut[pix] = make_hipFloatComplex(0., 0.);
        return;
    }

    // in range, continue

    // evaluate the doppler phase at the secondary coordinate
    double dop = evalPolyAt(inData.dopPoly, idxi+1+ao, idxj+1+ro);

    // phase corrections to be added later
    double ph = (dop * fraca) + evalPolyAt(inData.rgCarrierPoly, idxi+1+ao, idxj+1+ro) +
        evalPolyAt(inData.azCarrierPoly, idxi+1+ao, idxj+1+ro);

    // if flatten
    if (ini[7] == 1)
        ph = ph + ((4.*(M_PI/ind[0]))*((ind[2]-ind[3])+(idxj*(ind[4]-ind[5]))+(ro*ind[4])))
            +((4.*M_PI*(ind[3]+(idxj*ind[5])))*((1./ind[1])-(1./ind[0])));

    ph = modulo_f(ph, 2.*M_PI);

    // temp variable to keep track of the interpolated value
    hipFloatComplex cval = make_hipFloatComplex(0.,0.);
    // get the indices in the sinfc_coef of the fractional parts
    int ifraca = int(fraca*SINC_SUB);
    int ifracr = int(fracr*SINC_SUB);

    // weight for sinfc interp coefficients
    float weightsum = 0.;

    // iterate over the interpolation zone, e.g.  [-3, 4] x [-3, 4] for SINC_LEN = 8
    for (int i=-SINC_HALF+1; i<=SINC_HALF; i++) {
        hipFloatComplex cdop = make_hipFloatComplex(cosf(i*dop), -sinf(i*dop));
        for (int j=-SINC_HALF+1; j<=SINC_HALF; j++) {
            float weight = inData.fintp[IDX1D(ifraca,SINC_HALF-i,SINC_LEN)]
                            *inData.fintp[IDX1D(ifracr,SINC_HALF-j,SINC_LEN)];
            // correct the doppler phase here
            hipFloatComplex cin = hipCmulf(inData.imgIn[IDX1D(i+ka,j+kr,ini[1])], cdop);
            cval = hipCaddf(cval, make_hipFloatComplex(hipCrealf(cin)*weight, hipCimagf(cin)*weight));
            weightsum += weight;
        }
    }
    // normalize
    cval = make_hipFloatComplex(hipCrealf(cval)/weightsum, hipCimagf(cval)/weightsum);
    // phase correction
    cval = hipCmulf(cval, make_hipFloatComplex(cosf(ph), sinf(ph)));
    // assign and return
    inData.imgOut[pix] = cval;
}

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//              CPU Helper Functions
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

double cpuSecond() {

    struct timeval tp;
    gettimeofday(&tp,NULL);
    return (double(tp.tv_sec) + double(tp.tv_usec)*1.e-6);
}

void checkKernelErrors() {

    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();

    if (errSync != hipSuccess) printf("\nSync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess) printf("\nAsync kernel error: %s\n", hipGetErrorString(errAsync));
}

// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
//              Main CPU Function
// * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *

void runGPUResamp(double *h_inpts_dbl, int *h_inpts_int, void *imgIn, void *imgOut,
                    float *residAz, float *residRg, double *azOffPoly, double *rgOffPoly,
                    double *dopPoly, double *azCarrierPoly, double *rgCarrierPoly, float *fintp)
{
    /* * * * * * * * * * * * * * * * * * * *
     * Input mapping -
     *
     *  Double 0 - wvl
     *  Double 1 - refwvl
     *  Double 2 - r0
     *  Double 3 - refr0
     *  Double 4 - slr
     *  Double 5 - refslr
     *
     *  Int 0 - inLength
     *  Int 1 - inWidth
     *  Int 2 - outWidth
     *  Int 3 - firstImageRow
     *  Int 4 - firstTileRow
     *  Int 5 - nRowsInBlock
     *  Int 6 - LINES_PER_TILE
     *  Int 7 - flatten
     *
     * * * * * * * * * * * * * * * * * * * */

    // Casting input/output images to native cuFloatComplex type from complex<float>
    hipFloatComplex *h_imgIn = (hipFloatComplex *)imgIn;
    hipFloatComplex *h_imgOut = (hipFloatComplex *)imgOut;

    // Create handles for device copies of inputs
    hipFloatComplex *d_imgIn, *d_imgOut;
    float *d_residAz, *d_residRg;
    double *d_azOffPoly, *d_rgOffPoly, *d_dopPoly, *d_azCarrierPoly, *d_rgCarrierPoly;
    float *d_fintp;

    double startRun, endRun, startKernel, endKernel;

    struct InputData inData;


    printf("\n  Initializing GPU ResampSlc\n");
    hipSetDevice(0);

    startRun = cpuSecond();

    printf("    Allocating initial memory... ");
    fflush(stdout);

    int nInPix = h_inpts_int[5] * h_inpts_int[1];
    int nOutPix = h_inpts_int[6] * h_inpts_int[2];
    int nResidAzPix = 0;
    if (residAz != 0) nResidAzPix = h_inpts_int[6] * h_inpts_int[2];
    int nResidRgPix = 0;
    if (residRg != 0) nResidRgPix = h_inpts_int[6] * h_inpts_int[2];
    int nAzOffPix = ((azOffPoly[0]+1) * (azOffPoly[1]+1)) + 6; // [0] and [1] of the Poly2d arrays hold the az/rg orders
    int nRgOffPix = ((rgOffPoly[0]+1) * (rgOffPoly[1]+1)) + 6;
    int nDopPix = ((dopPoly[0]+1) * (dopPoly[1]+1)) + 6;
    int nAzCarryPix = ((azCarrierPoly[0]+1) * (azCarrierPoly[1]+1)) + 6;
    int nRgCarryPix = ((rgCarrierPoly[0]+1) * (rgCarrierPoly[1]+1)) + 6;

    size_t nb_in = nInPix * sizeof(hipFloatComplex);
    size_t nb_out = nOutPix * sizeof(hipFloatComplex);
    size_t nb_rsdAz = nResidAzPix * sizeof(float);
    size_t nb_rsdRg = nResidRgPix * sizeof(float);
    size_t nb_azOff = nAzOffPix * sizeof(double);
    size_t nb_rgOff = nRgOffPix * sizeof(double);
    size_t nb_dop = nDopPix * sizeof(double);
    size_t nb_azCarry = nAzCarryPix * sizeof(double);
    size_t nb_rgCarry = nRgCarryPix * sizeof(double);

    hipMalloc((hipFloatComplex**)&d_imgIn, nb_in);
    hipMalloc((hipFloatComplex**)&d_imgOut, nb_out);
    if (residAz != 0) hipMalloc((float**)&d_residAz, nb_rsdAz);
    if (residRg != 0) hipMalloc((float**)&d_residRg, nb_rsdRg);
    hipMalloc((double**)&d_azOffPoly, nb_azOff);
    hipMalloc((double**)&d_rgOffPoly, nb_rgOff);
    hipMalloc((double**)&d_dopPoly, nb_dop);
    hipMalloc((double**)&d_azCarrierPoly, nb_azCarry);
    hipMalloc((double**)&d_rgCarrierPoly, nb_rgCarry);
    hipMalloc((float**)&d_fintp, (SINC_LEN*SINC_SUB*sizeof(float)));

    printf("Done.\n    Copying data to GPU... ");
    fflush(stdout);

    startKernel = cpuSecond();

    hipMemcpy(d_imgIn, h_imgIn, nb_in, hipMemcpyHostToDevice);
    if (residAz != 0) hipMemcpy(d_residAz, residAz, nb_rsdAz, hipMemcpyHostToDevice);
    if (residRg != 0) hipMemcpy(d_residRg, residRg, nb_rsdRg, hipMemcpyHostToDevice);
    hipMemcpy(d_azOffPoly, azOffPoly, nb_azOff, hipMemcpyHostToDevice);
    hipMemcpy(d_rgOffPoly, rgOffPoly, nb_rgOff, hipMemcpyHostToDevice);
    hipMemcpy(d_dopPoly, dopPoly, nb_dop, hipMemcpyHostToDevice);
    hipMemcpy(d_azCarrierPoly, azCarrierPoly, nb_azCarry, hipMemcpyHostToDevice);
    hipMemcpy(d_rgCarrierPoly, rgCarrierPoly, nb_rgCarry, hipMemcpyHostToDevice);
    hipMemcpy(d_fintp, fintp, (SINC_LEN*SINC_SUB*sizeof(float)), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(ind), h_inpts_dbl, (6*sizeof(double)));
    hipMemcpyToSymbol(HIP_SYMBOL(ini), h_inpts_int, (8*sizeof(int)));

    hipMemset(d_imgOut, 0, nb_out);

    endKernel = cpuSecond();

    printf("Done. (%f s.)\n", (endKernel-startKernel));


    printf("    Running GPU ResampSlc... ");
    fflush(stdout);

    startKernel = cpuSecond();

    inData.imgIn = d_imgIn;
    inData.imgOut = d_imgOut;
    inData.residAz = 0;
    if (residAz != 0) inData.residAz = d_residAz;
    inData.residRg = 0;
    if (residRg != 0) inData.residRg = d_residRg;
    inData.azOffPoly = d_azOffPoly;
    inData.rgOffPoly = d_rgOffPoly;
    inData.dopPoly = d_dopPoly;
    inData.azCarrierPoly = d_azCarrierPoly;
    inData.rgCarrierPoly = d_rgCarrierPoly;
    inData.fintp = d_fintp;

   // remove carriers from the input image
    int threads = 1024;
    int blocks = (nInPix + threads-1) / threads;
    removeCarrier<<<blocks, threads>>>(inData);
    checkKernelErrors();
    // resample
    blocks = (nOutPix + threads -1) / threads;
    GPUResamp <<<blocks, threads>>>(inData);
    checkKernelErrors();

    endKernel = cpuSecond();

    printf("Done. (%f s.)\n", (endKernel-startKernel));

    printf("    Copying memory back to host... ");
    fflush(stdout);

    startKernel = cpuSecond();

    hipMemcpy(h_imgOut, d_imgOut, nb_out, hipMemcpyDeviceToHost);

    endKernel = cpuSecond();
    endRun = cpuSecond();

    printf("Done. (%f s.)\n", (endKernel-startKernel));
    printf("    Finished GPU ResampSlc in %f s.\n", (endRun-startRun));
    printf("    Cleaning device memory and returning to main ResampSlc function...\n");

    hipFree(d_imgIn);
    hipFree(d_imgOut);
    if (residAz != 0) hipFree(d_residAz);
    if (residRg != 0) hipFree(d_residRg);
    hipFree(d_azOffPoly);
    hipFree(d_rgOffPoly);
    hipFree(d_dopPoly);
    hipFree(d_azCarrierPoly);
    hipFree(d_rgCarrierPoly);
    hipFree(d_fintp);
    hipDeviceReset();

    printf("  Exiting GPU ResampSlc\n\n");
}
